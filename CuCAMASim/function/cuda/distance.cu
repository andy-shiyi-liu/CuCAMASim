#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "util/data.h"
#include "function/cuda/distance.cuh"

__global__ void helloWorld(){
    printf("Hello World from GPU!\n");
}

double* rangeQueryPairwise(ACAMArray *camArray, QueryData *queryData){
    printf("in Range Query Pairwise\n");
    throw std::runtime_error("NotImplementedError: Range distance is not implemented yet");
    return (double*)nullptr;
}

double* softRangePairwise(ACAMArray *camArray, QueryData *queryData){
    printf("in SoftRange Pairwise\n");
    throw std::runtime_error("NotImplementedError: Soft range distance is not implemented yet");
    return (double*)nullptr;
}
