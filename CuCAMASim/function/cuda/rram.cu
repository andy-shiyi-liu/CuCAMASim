#include "hip/hip_runtime.h"
#include <stdio.h>

#include <cstring>

#include "function/cuda/rram.cuh"
#include "function/cuda/util.cuh"
#include "util/consts.h"

enum RRAMNoiseType {
  GAUSSIAN,
  BOUNDED_GAUSSIAN,
  G_DEPENDENT,
};

// use newton's method to solve conductance from Vbd
__device__ inline double solveConductanceFromVbd(double Vbd,
                                                 RRAMCellType type) {
  double x = RRAM_STARTPOINT;
  double f = RRAMConduct2Vbd(x, type) - Vbd;
  double df = d_RRAMConduct2Vbd(x, type);
  for (uint64_t i = 0; fabs(f) > RRAM_TOLERANCE && i < RRAM_MAX_ITER; i++) {
    x = x - f / df;
    f = RRAMConduct2Vbd(x, type) - Vbd;
    df = d_RRAMConduct2Vbd(x, type);
  }
  if (fabs(f) > RRAM_TOLERANCE){
    printf("\033[0;31mERROR: Newton's method failed to converge\033[0m\n");
  }
  return x;
};

__global__ void Vbd2conductance(double *array, const CAMArrayDim camDim,
                                const RRAMCellType cellType) {
  getIx;
  getIy;

  uint32_t nx = camDim.nCols;
  uint32_t ny = camDim.nRows;

  outOfRangeReturn2D;
  uint32_t rowIdx = iy;
  uint32_t colIdx = ix;

  double lowerBdVbd = array[getCamIdx(rowIdx, colIdx, 0, camDim)];
  array[getCamIdx(rowIdx, colIdx, 0, camDim)] =
      solveConductanceFromVbd(lowerBdVbd, cellType);
  double upperBdVbd = array[getCamIdx(rowIdx, colIdx, 1, camDim)];
  array[getCamIdx(rowIdx, colIdx, 1, camDim)] =
      solveConductanceFromVbd(upperBdVbd, cellType);
};

__global__ void conductance2Vbd(double *array, const CAMArrayDim camDim,
                                const RRAMCellType cellType) {
  assert(camDim.nBoundaries == 2);

  uint32_t nx = camDim.nCols;
  uint32_t ny = camDim.nRows;
  getIx;
  getIy;
  outOfRangeReturn2D;
  uint32_t rowIdx = iy;
  uint32_t colIdx = ix;

  double lowerBdConductance = array[getCamIdx(rowIdx, colIdx, 0, camDim)];
  array[getCamIdx(rowIdx, colIdx, 0, camDim)] =
      RRAMConduct2Vbd(lowerBdConductance, cellType);
  double upperBdConductance = array[getCamIdx(rowIdx, colIdx, 1, camDim)];
  array[getCamIdx(rowIdx, colIdx, 1, camDim)] =
      RRAMConduct2Vbd(upperBdConductance, cellType);
};

__global__ void addRRAMVariation(double *array, uint32_t nRows, uint32_t nCols,
                                 uint32_t nBoundaries, RRAMCellType cellType,
                                 RRAMNoiseType noiseType) {
  // printf("in addRRAMVariation()\n");
  assert(nBoundaries == 2);
};

void addRRAMNoise(WriteNoise *writeNoise, ACAMArray *array) {
  assert(writeNoise->getNoiseConfig()->device == "RRAM");
  assert(writeNoise->getHasNoise());

  // get info
  uint32_t nRows = array->getNRows();
  uint32_t nCols = array->getNCols();
  uint32_t nBoundaries = array->getDim().nBoundaries;
  assert(nBoundaries == 2);
  std::string cellType = writeNoise->getCellDesign();
  std::map<std::string, std::map<std::string, std::string>> noiseType =
      writeNoise->getNoiseType();

  RRAMCellType cellTypeCUDA = INVALID_RRAM_CELL_TYPE;
  if (cellType == "6T2M") {
    cellTypeCUDA = CELL_6T2M;
  } else if (cellType == "8T2M") {
    cellTypeCUDA = CELL_8T2M;
  } else {
    throw std::runtime_error("Invalid RRAM cell type");
  }

  // copy data
  uint64_t nByte = nRows * nCols * nBoundaries * sizeof(double);
  double *camRawData_h = array->getData(FOR_CUDA_MEM_CPY);
  double *camRawData_d;
  CHECK(hipMalloc(&camRawData_d, nByte));
  CHECK(hipMemcpy(camRawData_d, camRawData_h, nByte, hipMemcpyHostToDevice));

  // grid block size
  const dim3 block(RRAM_NOISE_THREAD_X, RRAM_NOISE_THREAD_Y);
  const dim3 grid((long long int)(nRows - 1) / block.x + 1,
                  (long long int)(nCols - 1) / block.y + 1);

  // cuda stream
  hipStream_t stream;
  CHECK(hipStreamCreate(&stream));

  // convert to conductance
  Vbd2conductance<<<grid, block, 0, stream>>>(camRawData_d, array->getDim(),
                                             cellTypeCUDA);

  // iterate through noise types
  for (auto &it : noiseType) {
    std::string noise = it.first;
    if (noise == "variation") {
      std::map<std::string, std::string> params = it.second;

      std::string noiseType = params["type"];

      if (noiseType == "bounded_gaussian") {
        RRAMNoiseType noiseTypeCUDA = GAUSSIAN;
        addRRAMVariation<<<grid, block, 0, stream>>>(camRawData_d, nRows, nCols,
                                                     nBoundaries, cellTypeCUDA,
                                                     noiseTypeCUDA);
      } else {
        throw std::runtime_error("Invalid variation type: " + noiseType);
      }
    } else {
      throw std::runtime_error("Invalid noise type: " + noise);
    }
  }

  // convert back to Vbd
  conductance2Vbd<<<grid, block, 0, stream>>>(camRawData_d, array->getDim(),
                                              cellTypeCUDA);

  // post process
  CHECK(hipStreamSynchronize(stream));
  CHECK(hipMemcpy(camRawData_h, camRawData_d, nByte, hipMemcpyDeviceToHost));
  CHECK(hipFree(camRawData_d));
  CHECK(hipStreamDestroy(stream));

  std::cerr << "\033[33mWARNING: addRRAMNoise() is still under development"
            << std::endl;
}