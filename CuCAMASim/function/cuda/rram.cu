#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#include <cstring>
#include <fstream>
#include <string>

#include "function/cuda/rram.cuh"
#include "function/cuda/util.cuh"
#include "util/consts.h"

// use newton's method to solve conductance from Vbd
__device__ inline double solveConductanceFromVbd(double Vbd,
                                                 RRAMCellType type) {
  double x = RRAM_STARTPOINT;
  double f = RRAMConduct2Vbd(x, type) - Vbd;
  double df = d_RRAMConduct2Vbd(x, type);
  for (uint64_t i = 0; fabs(f) / df >= 0.1 * RRAM_TOLERANCE && i < RRAM_MAX_ITER; i++) {
    x = x - f / df;
    f = RRAMConduct2Vbd(x, type) - Vbd;
    df = d_RRAMConduct2Vbd(x, type);
  }
  if (fabs(f) > RRAM_TOLERANCE) {
    printf("\033[0;31mERROR: Newton's method failed to converge!\033[0m\n");
  }
  return x;
};

__global__ void Vbd2conductance(double *array, const CAMArrayDim camDim,
                                const RRAMCellType cellType) {
  getIx;
  getIy;

  uint32_t nx = camDim.nCols;
  uint32_t ny = camDim.nRows;

  outOfRangeReturn2D;
  uint32_t rowIdx = iy;
  uint32_t colIdx = ix;

  double lowerBdVbd = array[getCamIdx(rowIdx, colIdx, 0, camDim)];
  array[getCamIdx(rowIdx, colIdx, 0, camDim)] =
      solveConductanceFromVbd(lowerBdVbd, cellType);
  double upperBdVbd = array[getCamIdx(rowIdx, colIdx, 1, camDim)];
  array[getCamIdx(rowIdx, colIdx, 1, camDim)] =
      solveConductanceFromVbd(upperBdVbd, cellType);
};

__global__ void conductance2Vbd(double *array, const CAMArrayDim camDim,
                                const RRAMCellType cellType) {
  assert(camDim.nBoundaries == 2);

  uint32_t nx = camDim.nCols;
  uint32_t ny = camDim.nRows;
  getIx;
  getIy;
  outOfRangeReturn2D;
  uint32_t rowIdx = iy;
  uint32_t colIdx = ix;

  double lowerBdConductance = array[getCamIdx(rowIdx, colIdx, 0, camDim)];
  array[getCamIdx(rowIdx, colIdx, 0, camDim)] =
      RRAMConduct2Vbd(lowerBdConductance, cellType);
  double upperBdConductance = array[getCamIdx(rowIdx, colIdx, 1, camDim)];
  array[getCamIdx(rowIdx, colIdx, 1, camDim)] =
      RRAMConduct2Vbd(upperBdConductance, cellType);
};

__global__ void addBoundedGaussianVariation(double *array,
                                            const CAMArrayDim camDim,
                                            const double stdDev,
                                            const double bound,
                                            const double minConductance,
                                            const double maxConductance) {
  // printf("in addBoundedGaussianVariation()\n");
  assert(camDim.nBoundaries == 2);
  assert(minConductance <= maxConductance);
  uint32_t nx = camDim.nCols;
  uint32_t ny = camDim.nRows;
  getIx;
  getIy;
  getIdx2D;
  outOfRangeReturn2D;
  uint32_t rowIdx = iy;
  uint32_t colIdx = ix;

  // Initialize the random number generator
  hiprandState state;
  hiprand_init(clock64(), idx, 0, &state);

  double noise = max(min(hiprand_normal_double(&state) * stdDev, bound), -bound);
  assert(noise >= -bound);
  assert(noise <= bound);
  double lowerBdConductance = max(
      min(array[getCamIdx(rowIdx, colIdx, 0, camDim)] + noise, maxConductance),
      minConductance);
  assert(lowerBdConductance >= minConductance);
  assert(lowerBdConductance <= maxConductance);
  array[getCamIdx(rowIdx, colIdx, 0, camDim)] = lowerBdConductance;

  noise = max(min(hiprand_normal_double(&state) * stdDev, bound), -bound);
  assert(noise >= -bound);
  assert(noise <= bound);
  double upperBdConductance = max(
      min(array[getCamIdx(rowIdx, colIdx, 1, camDim)] + noise, maxConductance),
      minConductance);
  assert(upperBdConductance >= minConductance);
  assert(upperBdConductance <= maxConductance);
  array[getCamIdx(rowIdx, colIdx, 1, camDim)] = upperBdConductance;
};

void addRRAMNoise(WriteNoise *writeNoise, ACAMArray *array) {
  assert(writeNoise->getNoiseConfig()->device == "RRAM");
  assert(writeNoise->getHasNoise());

  // get info
  uint32_t nRows = array->getNRows();
  uint32_t nCols = array->getNCols();
  assert(nRows != 0 && nCols != 0);
  uint32_t nBoundaries = array->getDim().nBoundaries;
  assert(nBoundaries == 2);
  std::string cellType = writeNoise->getCellDesign();
  std::map<std::string, std::map<std::string, std::string>> noiseType =
      writeNoise->getNoiseType();

  RRAMCellType cellTypeCUDA = INVALID_RRAM_CELL_TYPE;
  if (cellType == "6T2M") {
    cellTypeCUDA = CELL_6T2M;
  } else if (cellType == "8T2M") {
    cellTypeCUDA = CELL_8T2M;
  } else {
    throw std::runtime_error("Invalid RRAM cell type");
  }

  // copy data
  uint64_t nByte = nRows * nCols * nBoundaries * sizeof(double);
  double *camRawData_h = array->getData(FOR_CUDA_MEM_CPY);
  double *camRawData_d;
  CHECK(hipMalloc(&camRawData_d, nByte));
  CHECK(hipMemcpy(camRawData_d, camRawData_h, nByte, hipMemcpyHostToDevice));

  // grid block size
  const dim3 block(RRAM_NOISE_THREAD_X, RRAM_NOISE_THREAD_Y);
  const dim3 grid((long long int)(nCols - 1) / block.x + 1,
                  (long long int)(nRows - 1) / block.y + 1);
  checkGridBlockSize(grid, block);

  // cuda stream
  hipStream_t stream;
  CHECK(hipStreamCreate(&stream));

  // convert to conductance
  Vbd2conductance<<<grid, block, 0, stream>>>(camRawData_d, array->getDim(),
                                              cellTypeCUDA);

  // iterate through noise types
  for (auto &it : noiseType) {
    std::string noise = it.first;
    if (noise == "variation") {
      std::map<std::string, std::string> params = it.second;

      std::string noiseType = params["type"];

      if (noiseType == "bounded_gaussian") {
        double stdDev = std::stod(params["stdDev"]);
        double bound = std::stod(params["bound"]);
        addBoundedGaussianVariation<<<grid, block, 0, stream>>>(
            camRawData_d, array->getDim(), stdDev, bound,
            writeNoise->getMinConductance(), writeNoise->getMaxConductance());
      } else {
        throw std::runtime_error("Invalid variation type: " + noiseType);
      }
    } else {
      throw std::runtime_error("Invalid noise type: " + noise);
    }
  }

  // // for debug
  // CHECK(hipStreamSynchronize(stream));
  // CHECK(hipMemcpy(camRawData_h, camRawData_d, nByte,
  // hipMemcpyDeviceToHost)); array->toCSV("/workspaces/CuCAMASim/1after.csv");

  // convert back to Vbd
  conductance2Vbd<<<grid, block, 0, stream>>>(camRawData_d, array->getDim(),
                                              cellTypeCUDA);

  // post process
  CHECK(hipStreamSynchronize(stream));
  CHECK(hipMemcpy(camRawData_h, camRawData_d, nByte, hipMemcpyDeviceToHost));
  CHECK(hipFree(camRawData_d));
  CHECK(hipStreamDestroy(stream));
}

__global__ void expandConductanceAll(double *array, const CAMArrayDim camDim,
                                     const double expandSize,
                                     const double minConductance,
                                     const double maxConductance) {
  assert(camDim.nBoundaries == 2);
  assert(minConductance <= maxConductance);
  uint32_t nx = camDim.nCols;
  uint32_t ny = camDim.nRows;
  getIx;
  getIy;
  outOfRangeReturn2D;
  uint32_t rowIdx = iy;
  uint32_t colIdx = ix;

  double lowerBdConductance = array[getCamIdx(rowIdx, colIdx, 0, camDim)];
  assert(lowerBdConductance >= minConductance - RRAM_TOLERANCE &&
         lowerBdConductance <= maxConductance + RRAM_TOLERANCE);
  lowerBdConductance = max(lowerBdConductance - expandSize, minConductance);
  array[getCamIdx(rowIdx, colIdx, 0, camDim)] = lowerBdConductance;

  double upperBdConductance = array[getCamIdx(rowIdx, colIdx, 1, camDim)];
  assert(upperBdConductance >= minConductance - RRAM_TOLERANCE &&
         upperBdConductance <= maxConductance + RRAM_TOLERANCE);
  upperBdConductance = min(upperBdConductance + expandSize, maxConductance);
  array[getCamIdx(rowIdx, colIdx, 1, camDim)] = upperBdConductance;
};

__global__ void expandDontCareOnly(double *array, const CAMArrayDim camDim,
                                   const double expandSize,
                                   const double minConvertConductance,
                                   const double maxConvertConductance,
                                   const double minConductance,
                                   const double maxConductance) {
  assert(camDim.nBoundaries == 2);
  assert(minConductance <= maxConductance);
  uint32_t nx = camDim.nCols;
  uint32_t ny = camDim.nRows;
  getIx;
  getIy;
  outOfRangeReturn2D;
  uint32_t rowIdx = iy;
  uint32_t colIdx = ix;

  double lowerBdConductance = array[getCamIdx(rowIdx, colIdx, 0, camDim)];
  assert(lowerBdConductance >= minConductance - RRAM_TOLERANCE &&
         lowerBdConductance <= maxConductance + RRAM_TOLERANCE);
  if (lowerBdConductance >= minConvertConductance - RRAM_TOLERANCE &&
      lowerBdConductance <= minConvertConductance + RRAM_TOLERANCE) {
    lowerBdConductance = max(lowerBdConductance - expandSize, minConductance);
    array[getCamIdx(rowIdx, colIdx, 0, camDim)] = lowerBdConductance;
  }

  double upperBdConductance = array[getCamIdx(rowIdx, colIdx, 1, camDim)];
  assert(upperBdConductance >= minConductance - RRAM_TOLERANCE &&
         upperBdConductance <= maxConductance + RRAM_TOLERANCE);
  if (upperBdConductance >= maxConvertConductance - RRAM_TOLERANCE &&
      upperBdConductance <= maxConvertConductance + RRAM_TOLERANCE) {
    upperBdConductance = min(upperBdConductance + expandSize, maxConductance);
    array[getCamIdx(rowIdx, colIdx, 1, camDim)] = upperBdConductance;
  }
};

void addRRAMNewMapping(Mapping *mapping, ACAMArray *array) {
  // get info
  uint32_t nRows = array->getNRows();
  uint32_t nCols = array->getNCols();
  assert(nRows != 0 && nCols != 0);
  uint32_t nBoundaries = array->getDim().nBoundaries;
  assert(nBoundaries == 2);
  std::string cellType = mapping->getCellConfig()->design;
  RRAMCellType cellTypeCUDA = INVALID_RRAM_CELL_TYPE;
  if (cellType == "6T2M") {
    cellTypeCUDA = CELL_6T2M;
  } else if (cellType == "8T2M") {
    cellTypeCUDA = CELL_8T2M;
  } else {
    throw std::runtime_error("Invalid RRAM cell type");
  }

  // copy data
  uint64_t nByte = nRows * nCols * nBoundaries * sizeof(double);
  double *camRawData_h = array->getData(FOR_CUDA_MEM_CPY);
  double *camRawData_d;
  CHECK(hipMalloc(&camRawData_d, nByte));
  CHECK(hipMemcpy(camRawData_d, camRawData_h, nByte, hipMemcpyHostToDevice));

  // grid block size
  const dim3 block(RRAM_NEWMAPPING_THREAD_X, RRAM_NEWMAPPING_THREAD_Y);
  const dim3 grid((long long int)(nCols - 1) / block.x + 1,
                  (long long int)(nRows - 1) / block.y + 1);
  checkGridBlockSize(grid, block);

  // cuda stream
  hipStream_t stream;
  CHECK(hipStreamCreate(&stream));

  // convert to conductance
  Vbd2conductance<<<grid, block, 0, stream>>>(camRawData_d, array->getDim(),
                                              cellTypeCUDA);

  // // for debug
  // CHECK(hipStreamSynchronize(stream));
  // CHECK(hipMemcpy(camRawData_h, camRawData_d, nByte,
  // hipMemcpyDeviceToHost));
  // array->toCSV("/workspaces/CuCAMASim/0before.csv");

  for (auto it : mapping->getMappingConfig()->strategies) {
    if (it.first == "expandConductanceAll") {
      std::map<std::string, std::string> params = it.second;
      assert((params["strategy"] == "fixed size") &&
             "Only support fixed size strategy now");
      assert(mapping->getCellConfig()->device == "RRAM");
      double expandSize = std::stod(params["expandSize"]);
      expandConductanceAll<<<grid, block, 0, stream>>>(
          camRawData_d, array->getDim(), expandSize,
          mapping->getCellConfig()->minConductance,
          mapping->getCellConfig()->maxConductance);
      std::cout << "added expandConductanceAll mapping strategy" << std::endl;
    } else if (it.first == "expandDontCareOnly") {
      std::map<std::string, std::string> params = it.second;
      assert((params["strategy"] == "fixed size") &&
             "Only support fixed size strategy now");
      assert(mapping->getCellConfig()->device == "RRAM");
      double expandSize = std::stod(params["expandSize"]);
      float minConvertConductance = std::stof(mapping->getMappingConfig()
                                                  ->strategies.at("N2VConvert")
                                                  .at("minConvertConductance"));
      float maxConvertConductance = std::stof(mapping->getMappingConfig()
                                                  ->strategies.at("N2VConvert")
                                                  .at("maxConvertConductance"));
      expandDontCareOnly<<<grid, block, 0, stream>>>(
          camRawData_d, array->getDim(), expandSize, minConvertConductance,
          maxConvertConductance, mapping->getCellConfig()->minConductance,
          mapping->getCellConfig()->maxConductance);
      std::cout << "added expandDontCareOnly mapping strategy" << std::endl;
    } else if (it.first == "N2VConvert") {
      // do nothing
    } else {
      throw std::runtime_error("Invalid mapping strategy: " + it.first);
    }
  }

  // convert back to Vbd
  conductance2Vbd<<<grid, block, 0, stream>>>(camRawData_d, array->getDim(),
                                              cellTypeCUDA);

  // post process
  CHECK(hipStreamSynchronize(stream));
  CHECK(hipMemcpy(camRawData_h, camRawData_d, nByte, hipMemcpyDeviceToHost));
  CHECK(hipFree(camRawData_d));
  CHECK(hipStreamDestroy(stream));
};