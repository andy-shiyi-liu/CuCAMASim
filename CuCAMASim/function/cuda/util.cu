#include <cassert>

#include "function/cuda/util.cuh"
#include "util/data.h"

void initDevice(int devNum) {
  int dev = devNum;
  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  printf("Using GPU device %d: %s\n", dev, deviceProp.name);
  CHECK(hipSetDevice(dev));
};