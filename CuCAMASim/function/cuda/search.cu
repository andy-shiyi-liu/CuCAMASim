#include <limits>

#include "function/cuda/distance.cuh"
#include "function/cuda/search.cuh"
#include "function/cuda/util.cuh"
#include "util/consts.h"

void CAMSearchCUDA(CAMSearch *CAMSearch, const CAMDataBase *camData,
                   const QueryData *queryData) {
  initDevice(0);
  const uint32_t nVectors = queryData->getNVectors();
  const uint32_t colCams = camData->getColCams();
  const uint32_t rowSize = camData->getRowSize();

  uint64_t matchIdxMaxCols = MAX_MATCHED_ROWS * colCams;
  uint32_t **matchIdx =
      new2DArray<uint32_t>(nVectors, matchIdxMaxCols, uint32_t(-1));
  double **matchIdxDist = new2DArray<double>(
      nVectors, matchIdxMaxCols, std::numeric_limits<double>::quiet_NaN());

  // 1. Search in multiple arrays
  for (uint32_t rowCamIdx = 0; rowCamIdx < camData->getRowCams(); rowCamIdx++) {
    for (uint32_t colCamIdx = 0; colCamIdx < camData->getColCams();
         colCamIdx++) {
      arraySearch(CAMSearch, camData, queryData, matchIdx, matchIdxDist,
                  rowCamIdx, colCamIdx);
    }
  }

  delete2DArray<uint32_t>(matchIdx, nVectors);
  delete2DArray<double>(matchIdxDist, nVectors);
  hipDeviceReset();
  std::cerr << "\033[33mWARNING: CAMSearchCUDA() is still under "
               "development\033[0m"
            << CAMSearch << camData << queryData << std::endl;
}

void arraySearch(const CAMSearch *CAMSearch, const CAMDataBase *camData,
                 const QueryData *queryData, uint32_t **matchIdx,
                 double **matchIdxDist, const uint32_t rowCamIdx,
                 const uint32_t colCamIdx) {
  uint32_t rowSize = camData->getRowSize(), colSize = camData->getColSize();

  double **distanceArray = new2DArray<double>(
      rowSize, colSize, std::numeric_limits<double>::quiet_NaN());

  if (CAMSearch->getDistType() == "euclidean") {
    throw std::runtime_error(
        "NotImplementedError: Euclidean distance is not implemented yet");
  } else if (CAMSearch->getDistType() == "manhattan") {
    throw std::runtime_error(
        "NotImplementedError: Manhattan distance is not implemented yet");
  } else if (CAMSearch->getDistType() == "hamming") {
    throw std::runtime_error(
        "NotImplementedError: Hamming distance is not implemented yet");
  } else if (CAMSearch->getDistType() == "innerproduct") {
    throw std::runtime_error(
        "NotImplementedError: Inner product distance is not implemented yet");
  } else if (CAMSearch->getDistType() == "range") {
    throw std::runtime_error(
    "NotImplementedError: Range distance is not implemented yet");
  } else if (CAMSearch->getDistType() == "softRange") {
    throw std::runtime_error(
        "NotImplementedError: Soft range distance is not implemented yet");
  } else {
    throw std::runtime_error("NotImplementedError: Unknown distance type");
  }

  std::cerr << "\033[33mWARNING: arraySearch() is still under "
               "development\033[0m"
            << CAMSearch << camData << queryData << std::endl;
}