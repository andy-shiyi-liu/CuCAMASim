#include "hip/hip_runtime.h"
#include <limits>

#include "function/cuda/distance.cuh"
#include "function/cuda/merge.cuh"
#include "function/cuda/search.cuh"
#include "function/cuda/sensing.cuh"
#include "function/cuda/util.cuh"
#include "util/consts.h"
#include "util/data.h"

void CAMSearchCUDA(CAMSearch *camSearch, const CAMDataBase *camData,
                   const QueryData *queryData, SimResult *simResult) {
  initDevice(0);
  const uint32_t nVectors = queryData->getNVectors();
  const uint32_t colCams = camData->getColCams();
  const uint32_t rowSize = camData->getRowSize();

  assert(colCams == camSearch->getColCams());
  assert(camData->getRowCams() == camSearch->getRowCams());

  uint64_t matchIdxMaxCols = MAX_MATCHED_ROWS * colCams;

  uint32_t *matchIdx_d;
  uint64_t nBytes = nVectors * matchIdxMaxCols * sizeof(uint32_t);
  CHECK(hipMalloc((void **)&matchIdx_d, nBytes));
  CHECK(hipMemset(
      matchIdx_d, 255,
      nBytes));  // set each bit in the array to 1, then if there is not a
                 // matched row, the index would be uint32_t(-1)

  double *matchIdxDist_d;
  nBytes = nVectors * matchIdxMaxCols * sizeof(double);
  CHECK(hipMalloc((void **)&matchIdxDist_d, nBytes));
  CHECK(
      hipMemset(matchIdxDist_d, 255,
                 nBytes));  // set each bit in the array to 1, then if there is
                            // not a matched row, the distance would be -nan

  // 1. Search in multiple arrays
  for (uint32_t rowCamIdx = 0; rowCamIdx < camData->getRowCams(); rowCamIdx++) {
    for (uint32_t colCamIdx = 0; colCamIdx < camData->getColCams();
         colCamIdx++) {
      arraySearch(camSearch, camData, queryData, matchIdx_d, matchIdxDist_d,
                  rowCamIdx, colCamIdx);
    }
  }

  // 2. Merge results from multiple arrays
  uint32_t *result_d;
  nBytes = nVectors * MAX_MATCHED_ROWS * sizeof(uint32_t);
  CHECK(hipMalloc((void **)&result_d, nBytes));
  CHECK(hipMemset(
      result_d, 255,
      nBytes));  // set each bit in the array to 1, then if there is not a
                 // matched row, the index would be uint32_t(-1)

  mergeIndices(camSearch, matchIdx_d, matchIdxDist_d, result_d, nVectors,
               colCams);

  hipDeviceReset();
  std::cerr << "\033[33mWARNING: CAMSearchCUDA() is still under "
               "development\033[0m"
            << camSearch << camData << queryData << std::endl;
}

void mergeIndices(const CAMSearch *camSearch, const uint32_t *matchIdx_d,
                  const double *matchIdxDist_d, uint32_t *result_d,
                  const uint32_t nVectors, const uint32_t colCams) {
  dim3 block4Merging(MERGING_THREAD_X);
  dim3 grid4Merging((nVectors - 1) / block4Merging.x +
                    1);  // we need #nVectors threads

  if (camSearch->getSearchScheme() == "exact") {
    exactMerge<<<grid4Merging, block4Merging>>>(matchIdx_d, matchIdxDist_d,
                                                result_d, nVectors, colCams);
  } else if (camSearch->getSearchScheme() == "knn") {
    throw std::runtime_error(
        "NotImplementedError: KNN sensing is not implemented yet");
  } else if (camSearch->getSearchScheme() == "threshold") {
    throw std::runtime_error(
        "NotImplementedError: Threshold sensing is not implemented yet");
  } else {
    throw std::runtime_error("NotImplementedError: Unknown merge scheme");
  }

  // for debug
  // export result_d to csv file
  uint32_t *result_h = new uint32_t[nVectors * MAX_MATCHED_ROWS * 1];
  CHECK(hipMemcpy(result_h, result_d,
                   nVectors * MAX_MATCHED_ROWS * 1 * sizeof(uint32_t),
                   hipMemcpyDeviceToHost));
  std::ofstream file5("/workspaces/CuCAMASim/result.csv");
  file5 << ",";
  for (uint32_t i = 0; i < MAX_MATCHED_ROWS * 1; i++) {
    file5 << i << ",";
  }
  file5 << std::endl;
  for (uint32_t i = 0; i < nVectors; i++) {
    file5 << i << ",";
    for (uint32_t j = 0; j < MAX_MATCHED_ROWS * 1; j++) {
      file5 << result_h[j + MAX_MATCHED_ROWS * 1 * i] << ",";
    }
    file5 << std::endl;
  }
  file5.close();

  std::cerr
      << "\033[33mWARNING: mergeIndices() is still under development\033[0m"
      << matchIdx_d << matchIdxDist_d << result_d << nVectors << colCams
      << std::endl;
}

// for each CAM subarray, search and give the matched index and distance
void arraySearch(const CAMSearch *camSearch, const CAMDataBase *camData,
                 const QueryData *queryData, uint32_t *matchIdx_d,
                 double *matchIdxDist_d, const uint32_t rowCamIdx,
                 const uint32_t colCamIdx) {
  // get and check data dimensions
  const uint32_t rowSize = camData->getRowSize(),
                 colSize = camData->getColSize(),
                 nVectors = queryData->getNVectors();
  const CAMArrayDim camDim = camData->at(rowCamIdx, colCamIdx)->getDim();
  const InputDataDim queryDim = queryData->at(colCamIdx)->getDim();

  assert(camDim.nCols == queryDim.nFeatures);
  assert(camDim.nCols == rowSize);
  assert(queryDim.nVectors == nVectors);
  assert(camDim.nCols == colSize);

  // init data for cuda kernel
  const double *rawCamData_h =
      camData->at(rowCamIdx, colCamIdx)->getData(FOR_CUDA_MEM_CPY);
  double *rawCamData_d;
  uint64_t nBytes =
      camDim.nRows * camDim.nCols * camDim.nBoundaries * sizeof(double);
  CHECK(hipMalloc((void **)&rawCamData_d, nBytes));
  CHECK(hipMemcpy(rawCamData_d, rawCamData_h, nBytes, hipMemcpyHostToDevice));

  const double *rawQueryData_h =
      queryData->at(colCamIdx)->getData(FOR_CUDA_MEM_CPY);
  double *rawQueryData_d;
  nBytes = nVectors * colSize * sizeof(double);
  CHECK(hipMalloc((void **)&rawQueryData_d, nBytes));
  CHECK(hipMemcpy(rawQueryData_d, rawQueryData_h, nBytes,
                   hipMemcpyHostToDevice));

  // cuda grid and block size
  nBytes = nVectors * rowSize * sizeof(double);
  double *distanceArray_d;
  CHECK(hipMalloc((void **)&distanceArray_d, nBytes));
  dim3 block4Dist(DIST_FUNC_THREAD_X, DIST_FUNC_THREAD_Y);
  dim3 grid4Dist((long long int)(rowSize - 1) / block4Dist.x + 1,
                 (long long int)(nVectors - 1) / block4Dist.y + 1);

  // create cuda stream for sequential execution of kernels
  hipStream_t stream;
  CHECK(hipStreamCreate(&stream));

  // 1. Calculate the distance matrix in the array
  if (camSearch->getDistType() == "euclidean") {
    throw std::runtime_error(
        "NotImplementedError: Euclidean distance is not implemented yet");
  } else if (camSearch->getDistType() == "manhattan") {
    throw std::runtime_error(
        "NotImplementedError: Manhattan distance is not implemented yet");
  } else if (camSearch->getDistType() == "hamming") {
    throw std::runtime_error(
        "NotImplementedError: Hamming distance is not implemented yet");
  } else if (camSearch->getDistType() == "innerproduct") {
    throw std::runtime_error(
        "NotImplementedError: Inner product distance is not implemented yet");
  } else if (camSearch->getDistType() == "range") {
    // throw std::runtime_error(
    //     "NotImplementedError: Range distance is not implemented yet");
    if (camDim.nBoundaries != 2) {
      throw std::runtime_error(
          "Range distance requires ACAM, with 2 boundaries per cell!");
    }
    rangeQueryPairwise<<<grid4Dist, block4Dist, 0, stream>>>(
        rawCamData_d, rawQueryData_d, distanceArray_d, camDim, queryDim);
  } else if (camSearch->getDistType() == "softRange") {
    throw std::runtime_error(
        "NotImplementedError: Soft range distance is not implemented yet");
  } else {
    throw std::runtime_error("NotImplementedError: Unknown distance type");
  }

  // 2. Find the output IDs of the array
  dim3 block4Sensing(SENSING_THREAD_X);
  dim3 grid4Sensing((nVectors - 1) / block4Sensing.x +
                    1);  // we need #nVectors threads
  uint32_t errorCode = 0, *errorCode_d = nullptr;
  CHECK(hipMalloc((void **)&errorCode_d, sizeof(uint32_t)));
  CHECK(hipMemcpy(errorCode_d, &errorCode, sizeof(uint32_t),
                   hipMemcpyHostToDevice));
  if (camSearch->getSensing() == "exact") {
    getArrayExactResults<<<grid4Sensing, block4Sensing, 0, stream>>>(
        distanceArray_d, matchIdx_d, matchIdxDist_d, camDim, queryDim,
        rowCamIdx, colCamIdx, camData->getColCams(), errorCode_d);
  } else if (camSearch->getSensing() == "best") {
    throw std::runtime_error(
        "NotImplementedError: Best sensing is not implemented yet");
  } else if (camSearch->getSensing() == "threshold") {
    throw std::runtime_error(
        "NotImplementedError: Threshold sensing is not implemented yet");
  } else {
    throw std::runtime_error("NotImplementedError: Unknown sensing type");
  }
  // check error code
  CHECK(hipMemcpy(&errorCode, errorCode_d, sizeof(uint32_t),
                   hipMemcpyDeviceToHost));
  switch (errorCode) {
    case 1:
      throw std::runtime_error("Error: more than " +
                               std::to_string(MAX_MATCHED_ROWS) +
                               " matched. Please increase MAX_MATCHED_ROWS in "
                               "<CuCAMASim dir>/include/util/consts.h");
  }
  // Synchronize stream
  hipStreamSynchronize(stream);
  // Destroy stream
  hipStreamDestroy(stream);

  // for debug
  double *distanceArray_h = new double[nVectors * rowSize];
  CHECK(hipMemcpy(distanceArray_h, distanceArray_d, nBytes,
                   hipMemcpyDeviceToHost));

  // export distanceArray_h to csv file
  std::ofstream file("/workspaces/CuCAMASim/distances.csv");
  file << ",";
  for (uint32_t i = 0; i < rowSize; i++) {
    file << i << ",";
  }
  file << std::endl;
  for (uint32_t i = 0; i < nVectors; i++) {
    file << i << ",";
    for (uint32_t j = 0; j < rowSize; j++) {
      file << distanceArray_h[i * rowSize + j] << ",";
    }
    file << std::endl;
  }
  file.close();

  // export rawCamData_h to csv file
  std::ofstream file2("/workspaces/CuCAMASim/rawCamData.csv");
  // print col2featureID as column name
  file2 << ",";
  for (uint32_t i = 0; i < colSize; i++) {
    file2 << "col_" << i << ",";
  }
  file2 << "classID" << std::endl;
  for (uint32_t i = 0; i < rowSize; i++) {
    file2 << "row_" << i << ",";
    for (uint32_t j = 0; j < colSize; j++) {
      uint64_t lowerBdIdx = 0 + 2 * (j + colSize * i);
      uint64_t upperBdIdx = 1 + 2 * (j + colSize * i);
      file2 << rawCamData_h[lowerBdIdx]
            << " < x <= " << rawCamData_h[upperBdIdx] << ",";
    }
    file2 << std::endl;
  }
  file2.close();

  camData->at(rowCamIdx, colCamIdx)
      ->toCSV("/workspaces/CuCAMASim/camArray.csv");

  // export rawQueryData_h to csv file
  std::ofstream file3("/workspaces/CuCAMASim/rawQueryData.csv");
  file3 << ",";
  for (uint32_t i = 0; i < colSize; i++) {
    file3 << i << ",";
  }
  file3 << std::endl;
  for (uint32_t i = 0; i < nVectors; i++) {
    file3 << i << ",";
    for (uint32_t j = 0; j < colSize; j++) {
      file3 << rawQueryData_h[i * colSize + j] << ",";
    }
    file3 << std::endl;
  }
  file3.close();

  // export matchIdx_d to csv file
  uint32_t *matchIdx_h =
      new uint32_t[nVectors * MAX_MATCHED_ROWS * camData->getColCams()];
  CHECK(hipMemcpy(
      matchIdx_h, matchIdx_d,
      nVectors * MAX_MATCHED_ROWS * camData->getColCams() * sizeof(uint32_t),
      hipMemcpyDeviceToHost));
  std::ofstream file4("/workspaces/CuCAMASim/matchIdx.csv");
  file4 << ",";
  for (uint32_t i = 0; i < MAX_MATCHED_ROWS * camData->getColCams(); i++) {
    file4 << i << ",";
  }
  file4 << std::endl;
  for (uint32_t i = 0; i < nVectors; i++) {
    file4 << i << ",";
    for (uint32_t j = 0; j < MAX_MATCHED_ROWS * camData->getColCams(); j++) {
      file4 << matchIdx_h[j + MAX_MATCHED_ROWS * camData->getColCams() * i]
            << ",";
    }
    file4 << std::endl;
  }
  file4.close();

  // export matchIdxDist_d to csv file
  double *matchIdxDist_h =
      new double[nVectors * MAX_MATCHED_ROWS * camData->getColCams()];
  CHECK(hipMemcpy(
      matchIdxDist_h, matchIdxDist_d,
      nVectors * MAX_MATCHED_ROWS * camData->getColCams() * sizeof(double),
      hipMemcpyDeviceToHost));
  std::ofstream file5("/workspaces/CuCAMASim/matchIdxDist.csv");
  file5 << ",";
  for (uint32_t i = 0; i < MAX_MATCHED_ROWS * camData->getColCams(); i++) {
    file5 << i << ",";
  }
  file5 << std::endl;
  for (uint32_t i = 0; i < nVectors; i++) {
    file5 << i << ",";
    for (uint32_t j = 0; j < MAX_MATCHED_ROWS * camData->getColCams(); j++) {
      file5 << matchIdxDist_h[j + MAX_MATCHED_ROWS * camData->getColCams() * i]
            << ",";
    }
    file5 << std::endl;
  }
  file5.close();

  // std::cerr << "\033[33mWARNING: arraySearch() is still under "
  //              "development\033[0m"
  //           << camSearch << camData << queryData << std::endl;
}